#include "hip/hip_runtime.h"
#include "cuda_sample.cuh"
#include "cuda_util.cuh"
#include "fv1/cuda_fv1_flow.cuh"
#include "fv1/cuda_fv1_solver.cuh"

namespace lis
{
namespace cuda
{

__global__ void sample
(
	NUMERIC_TYPE* H,
	NUMERIC_TYPE* HU,
	NUMERIC_TYPE* HV,
	SampleBuffer sample_buf,
	NUMERIC_TYPE t
)
{
	sample_buf.time[cuda::sample_buf_idx] = t;

	for (int i=0; i<cuda::sample_points.count; i++)
	{
		int buf_offset = cuda::sample_buf_idx*cuda::sample_points.count;

		if (cuda::sample_points.inside_domain[i])
		{
			NUMERIC_TYPE Hval = H[cuda::sample_points.idx[i]];
			sample_buf.H[buf_offset + i] = Hval;

			if (Hval > cuda::solver_params.DepthThresh)
			{
				NUMERIC_TYPE HUval = HU[cuda::sample_points.idx[i]];
				NUMERIC_TYPE HVval = HV[cuda::sample_points.idx[i]];

				NUMERIC_TYPE speed = sqrt(HUval/Hval * HUval/Hval
						+ HVval/Hval * HVval/Hval);
				sample_buf.speed[buf_offset + i] = speed;
			}
		}
	}

	cuda::sample_buf_idx++;
}

}
}

lis::cuda::Sampler::Sampler
(
	SamplePoints& d_sample_points,
	int& sample_buf_idx,
	int verbose
)
:
d_sample_points(d_sample_points),
sample_buf_idx(sample_buf_idx),
stage_file(sample_points),
gauge_file(sample_points),
verbose(verbose)
{}

void lis::cuda::Sampler::load_sample_points
(
	const char* filename,
	Geometry& geometry,
	int pitch,
	int offset
)
{
	lis::Sample::initialise(sample_points, filename, geometry, pitch, offset,
			verbose);
	initialise_sample_points();

	allocate_pinned(sample_buf, sample_points.count);
	allocate_device(d_sample_buf, sample_points.count);

	active = true;
}

void lis::cuda::Sampler::open_stage_file
(
	const char* filename
)
{
	stage_file.open(filename);	
}

void lis::cuda::Sampler::open_gauge_file
(
	const char* filename
)
{
	gauge_file.open(filename);	

	write_speed = true;
}

void lis::cuda::Sampler::write_stage_header
(
	NUMERIC_TYPE* DEM,
	const char* sample_points_filename
)
{
	stage_file.write_header(DEM, sample_points_filename);
}

void lis::cuda::Sampler::write_gauge_header
(
	NUMERIC_TYPE* DEM,
	const char* sample_points_filename
)
{
	gauge_file.write_header(DEM, sample_points_filename);
}

void lis::cuda::Sampler::sample
(
	NUMERIC_TYPE* H,
	NUMERIC_TYPE* HU,
	NUMERIC_TYPE* HV,
	NUMERIC_TYPE t
)
{
	if (active) lis::cuda::sample<<<1, 1>>>(H, HU, HV, d_sample_buf, t);
}

void lis::cuda::Sampler::write_if_buffer_full()
{
	if (active && buffer_full())
	{
		if (verbose == ON) printf("SampleBuffer full: flushing to disk\n");
		copy_buffer();
		stage_file.write(sample_buf, sample_buf_idx);
		if (write_speed) gauge_file.write(sample_buf, sample_buf_idx);
		sample_buf_idx = 0;
	}
}

void lis::cuda::Sampler::write()
{
	if (active)
	{
		if (verbose == ON) printf("Flushing SampleBuffer to disk\n");
		copy_buffer();
		stage_file.write(sample_buf, sample_buf_idx);
		if (write_speed) gauge_file.write(sample_buf, sample_buf_idx);
	}
}

bool lis::cuda::Sampler::buffer_full()
{
	return sample_buf_idx == sample_buf.size;
}

void lis::cuda::Sampler::copy_buffer()
{
	SampleBuffer& dst = sample_buf;
	SampleBuffer& src = d_sample_buf;

	cuda::copy(dst.time, src.time, sample_buf_idx*sizeof(NUMERIC_TYPE));
	cuda::copy(dst.H, src.H,
			sample_buf_idx*sample_points.count*sizeof(NUMERIC_TYPE));
	cuda::copy(dst.speed, src.speed,
			sample_buf_idx*sample_points.count*sizeof(NUMERIC_TYPE));
}

void lis::cuda::Sampler::initialise_sample_points()
{
	SamplePoints& d_dst = d_sample_points;
	SamplePoints& h_src = sample_points;

	SamplePoints temp;
	int count = temp.count = h_src.count;

	temp.x = static_cast<NUMERIC_TYPE*>(
			malloc_device(count*sizeof(NUMERIC_TYPE)));
	temp.y = static_cast<NUMERIC_TYPE*>(
			malloc_device(count*sizeof(NUMERIC_TYPE)));
	temp.idx = static_cast<int*>(malloc_device(count*sizeof(int)));
	temp.inside_domain = static_cast<bool*>(malloc_device(count*sizeof(bool)));

	cuda::copy(temp.x, h_src.x, count*sizeof(NUMERIC_TYPE));
	cuda::copy(temp.y, h_src.y, count*sizeof(NUMERIC_TYPE));
	cuda::copy(temp.idx, h_src.idx, count*sizeof(int));
	cuda::copy(temp.inside_domain, h_src.inside_domain, count*sizeof(bool));

	copy_to_symbol(d_dst, &temp, sizeof(SamplePoints));
}

void lis::cuda::Sampler::free
(
	SamplePoints& d_sample_points
)
{
	cuda::free_device(d_sample_points.x);
	cuda::free_device(d_sample_points.y);
	cuda::free_device(d_sample_points.idx);
	cuda::free_device(d_sample_points.inside_domain);
}

void lis::cuda::Sampler::allocate_pinned
(
	SampleBuffer& buf,
	int points,
	int size
)
{
	buf.size = size;
	buf.time = static_cast<NUMERIC_TYPE*>(
			malloc_pinned(size*sizeof(NUMERIC_TYPE)));
	buf.H = static_cast<NUMERIC_TYPE*>(
			malloc_pinned(size*points*sizeof(NUMERIC_TYPE)));
	buf.speed = static_cast<NUMERIC_TYPE*>(
			malloc_pinned(size*points*sizeof(NUMERIC_TYPE)));
}

void lis::cuda::Sampler::allocate_device
(
	SampleBuffer& buf,
	int points,
	int size
)
{
	buf.size = size;
	buf.time = static_cast<NUMERIC_TYPE*>(
			malloc_device(size*sizeof(NUMERIC_TYPE)));
	buf.H = static_cast<NUMERIC_TYPE*>(
			malloc_device(size*points*sizeof(NUMERIC_TYPE)));
	buf.speed = static_cast<NUMERIC_TYPE*>(
			malloc_device(size*points*sizeof(NUMERIC_TYPE)));
}

void lis::cuda::Sampler::free_pinned
(
	SampleBuffer& buf
)
{
	cuda::free_pinned(buf.time);
	cuda::free_pinned(buf.H);
	cuda::free_pinned(buf.speed);
}

void lis::cuda::Sampler::free_device
(
	SampleBuffer& buf
)
{
	cuda::free_device(buf.time);
	cuda::free_device(buf.H);
	cuda::free_device(buf.speed);
}

lis::cuda::Sampler::~Sampler()
{
	if (active)
	{
		lis::Sample::free(sample_points);
		free(d_sample_points);
		free_pinned(sample_buf);
		free_device(d_sample_buf);
	}
}
